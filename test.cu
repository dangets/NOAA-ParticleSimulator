#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <cstddef>
#include <cstdlib>
#include <vector>


struct Particle {
    float pos_x;
    float pos_y;
    float pos_z;

    // bool has_deposited;
    // source_id
    // age
    // pollutant_type, mass, etc...

    Particle() { }
    Particle(float x, float y, float z) :
        pos_x(x), pos_y(y), pos_z(z) { }

    std::string toString() {
        std::ostringstream os;
        os << std::fixed << std::setprecision(3);
        os << "x:" << pos_x << "  ";
        os << "y:" << pos_y << "  ";
        os << "z:" << pos_z;
        return os.str();
    }
};


//struct Particles {
//    float4 *position;
//    float4 *velocity;
//
//    // type
//    // source_id
//    // age
//    // ...
//    size_t length;
//
//    Particles(int length_) : length(length_) { }
//    ~Particles() { }
//};


struct DataBrick {
    float x_start;
    float x_step;
    size_t x_length;

    float y_start;
    float y_step;
    size_t y_length;

    float z_start;
    float z_step;
    size_t z_length;

    time_t t_start;
    time_t t_step;
    size_t t_length;

    float *data;
};


float DataBrickGetValue(float x, float y, float z, time_t t)
{
    float diff = x - x_start;
    float quot = diff / x_step;
    size_t pos = (size_t)quot;
    float rem = quot - pos;

}



// particle positions
Particle *dev_particles;

// meteo grid info
size_t dev_grid_num_x;
size_t dev_grid_num_y;
size_t dev_grid_num_z;

// wind velocities
texture<float> texWindU;    // [t][z][y][x]
texture<float> texWindV;
texture<float> texWindW;

float *dev_windU;
float *dev_windV;
float *dev_windW;


//struct ParticleSource {
//    int id;
//
//    float pos_x;
//    float pos_y;
//    float pos_z;
//};



__global__
void advect_particles(Particle *particles, size_t num_particles,
        size_t grid_num_x, size_t grid_num_y, size_t grid_num_z)
{
    size_t i = threadIdx.x;
    if (i >= num_particles)
        return;

    size_t pos_offset =
        particles[i].pos_x +
        particles[i].pos_y * grid_num_x +
        particles[i].pos_z * grid_num_y * grid_num_z;

    float vel_u = tex1Dfetch(texWindU, pos_offset);
    float vel_v = tex1Dfetch(texWindV, pos_offset);
    float vel_w = tex1Dfetch(texWindW, pos_offset);

    particles[i].pos_x += vel_u;
    particles[i].pos_y += vel_v;
    particles[i].pos_z += vel_w;

    if (particles[i].pos_x < 0)
        particles[i].pos_x = 0;
    if (particles[i].pos_x >= grid_num_x)
        particles[i].pos_x = grid_num_x - 1;

    if (particles[i].pos_y < 0)
        particles[i].pos_y = 0;
    if (particles[i].pos_y >= grid_num_y)
        particles[i].pos_y = grid_num_y - 1;

    if (particles[i].pos_z < 0)
        particles[i].pos_z = 0;
    if (particles[i].pos_z >= grid_num_z)
        particles[i].pos_z = grid_num_z - 1;
}


void print_particles(Particle *particles, size_t num)
{
    for (int i=0; i<num; i++) {
        std::cout << particles[i].toString() << std::endl;
    }
    std::cout << "-----------------" << std::endl;
}


void print_dev_particles(size_t num_particles)
{
    size_t num_bytes = num_particles * sizeof(Particle);

    Particle *tmp = (Particle *)malloc(num_bytes);
    hipMemcpy(tmp, dev_particles, num_bytes, hipMemcpyDeviceToHost);
    print_particles(tmp, num_particles);
    free(tmp);
}


void init_particles(size_t n, size_t x, size_t y, size_t z)
{
    size_t num_bytes = n * sizeof(Particle);

    // allocate particle array
    hipMalloc((void **)&dev_particles, num_bytes);

    // allocate an array on the host to copy to the device
    Particle *tmp = (Particle *)malloc(num_bytes);
    for (int i=0; i<n; i++) {
        tmp[i].pos_x = x + i;
        tmp[i].pos_y = y + i;
        tmp[i].pos_z = z + i;
    }

    hipMemcpy(dev_particles, tmp, num_bytes, hipMemcpyHostToDevice);

    free(tmp);
}


void init_grid(size_t num_x, size_t num_y, size_t num_z, size_t num_t)
{
    size_t num_cells = num_x * num_y * num_z * num_t;
    size_t num_bytes = num_cells * sizeof(float);

    //std::cout << "num_bytes: " << num_bytes << std::endl;

    // allocate the device arrays
    hipMalloc((void **)&dev_windU, num_bytes);
    hipMalloc((void **)&dev_windV, num_bytes);
    hipMalloc((void **)&dev_windW, num_bytes);

    // bind the texture references to the arrays
    hipBindTexture(NULL, texWindU, dev_windU, num_bytes);
    hipBindTexture(NULL, texWindV, dev_windV, num_bytes);
    hipBindTexture(NULL, texWindW, dev_windW, num_bytes);

    // initialize the device arrays with random data
    float *tmp_data = (float *)malloc(num_bytes);
    for (int i=0; i<num_cells; i++) {
        tmp_data[i] = (float)drand48();
    }

    hipMemcpy(dev_windU, tmp_data, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_windV, tmp_data, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_windW, tmp_data, num_bytes, hipMemcpyHostToDevice);

    free(tmp_data);
}


void cleanup()
{
    // cleanup particles
    hipFree(dev_particles);

    // cleanup grid
    hipUnbindTexture(texWindU);
    hipUnbindTexture(texWindV);
    hipUnbindTexture(texWindW);
    hipFree(dev_windU);
    hipFree(dev_windV);
    hipFree(dev_windW);
}


int main(int argc, char *argv[])
{
    // init settings/configuration
    // init meteorology grid

    init_grid(256, 256, 16, 32);
    init_particles(10, 1, 2, 3);

    print_dev_particles(10);
    advect_particles<<<1, 16>>>(dev_particles, 10, 256, 256, 16);
    print_dev_particles(10);
    advect_particles<<<1, 16>>>(dev_particles, 10, 256, 256, 16);
    print_dev_particles(10);
    advect_particles<<<1, 16>>>(dev_particles, 10, 256, 256, 16);
    print_dev_particles(10);

    //Grid g = Grid(512, 512, 128);
    //for (int t=0; t<10; t++) {
    //    advect_particle(p);
    //    std::cout << p.toString() << std::endl;
    //}

    cleanup();

//    for (int timestep_i=0; timestep_i < num_timesteps; timestep_i++) {
//        // introduce new particles into the system
//
//        for (int pollutant_i=0; pollutant_i < num_pollutants; pollutant_i++) {
//            // skip terminated particles
//
//            // advect particle
//
//            // determine terrain height
//
//            // dry, wet, decay routines
//            // surface water transport
//        }
//    }

    return 0;
}

