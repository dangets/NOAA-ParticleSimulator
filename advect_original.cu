#include "hip/hip_runtime.h"
/*
   Author: Danny George
   High Performance Simulation Laboratory
   Boise State University
 
   Permission is hereby granted, free of charge, to any person obtaining a copy of
   this software and associated documentation files (the "Software"), to deal in
   the Software without restriction, including without limitation the rights to
   use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
   of the Software, and to permit persons to whom the Software is furnished to do
   so, subject to the following conditions:

   The above copyright notice and this permission notice shall be included in all
   copies or substantial portions of the Software.

   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
   IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
   AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
   LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
   OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
   SOFTWARE. */

#include "advect_original.cuh"

namespace {

struct AdvectOriginalFunctor {
    template <typename WindData>
    AdvectOriginalFunctor(const WindData &wd, const int &t_)
        : u(thrust::raw_pointer_cast(&wd.u[0])),
            v(thrust::raw_pointer_cast(&wd.v[0])),
            w(thrust::raw_pointer_cast(&wd.w[0])),
            t(t_),
            num_x(wd.shape.x()), num_y(wd.shape.y()), num_z(wd.shape.z()), num_t(wd.shape.t())
    { }

    const float *u;         // pointer to WindData's 'u' data
    const float *v;         // pointer to WindData's 'v' data
    const float *w;         // pointer to WindData's 'w' data

    const int t;

    const size_t num_x;
    const size_t num_y;
    const size_t num_z;
    const size_t num_t;

    __host__ __device__
    inline size_t get_index(size_t x, size_t y, size_t z, size_t t) const {
        // TODO: hardcoded t to 0 for development ~v~~~~~~~~~
        return x + y * num_x + z * num_y * num_x + 0 * num_x * num_y * num_z;
    }

    __host__ __device__
    float3 get_velocity_border0(float x, float y, float z, float t) const {
        float3 ret = make_float3(0.0f, 0.0f, 0.0f);
        size_t x0, x1;
        size_t y0, y1;
        size_t z0, z1;

        // border logic ----------
        if (x <= 0) {
            return ret;
        } else if (x >= num_x-1) {
            return ret;
        } else {
            x0 = (size_t)x;
            x1 = x+1;
        }

        if (y <= 0) {
            return ret;
        } else if (y >= num_y-1) {
            return ret;
        } else {
            y0 = (size_t)y;
            y1 = y+1;
        }

        if (z <= 0) {
            return ret;
        } else if (z >= num_z-1) {
            return ret;
        } else {
            z0 = (size_t)z;
            z1 = z+1;
        }

        // distance from actual point to sampled point index
        float x_d = x - x0;
        float y_d = y - y0;
        float z_d = z - z0;

        size_t i000 = get_index(x0, y0, z0, t);
        size_t i100 = get_index(x1, y1, z0, t);
        size_t i010 = get_index(x0, y1, z0, t);
        size_t i110 = get_index(x1, y1, z0, t);
        size_t i001 = get_index(x0, y0, z1, t);
        size_t i101 = get_index(x1, y0, z1, t);
        size_t i011 = get_index(x0, y1, z1, t);
        size_t i111 = get_index(x1, y1, z1, t);

        float c00 = u[i000] * (1 - x_d) + u[i100] * x_d;
        float c10 = u[i010] * (1 - x_d) + u[i110] * x_d;
        float c01 = u[i001] * (1 - x_d) + u[i101] * x_d;
        float c11 = u[i011] * (1 - x_d) + u[i111] * x_d;

        float c0 = c00 * (1 - y_d) + c10 * y_d;
        float c1 = c01 * (1 - y_d) + c11 * y_d;

        ret.x = (c0 * (1 - z_d) + c1 * z_d);

        c00 = v[i000] * (1 - x_d) + v[i100] * x_d;
        c10 = v[i010] * (1 - x_d) + v[i110] * x_d;
        c01 = v[i001] * (1 - x_d) + v[i101] * x_d;
        c11 = v[i011] * (1 - x_d) + v[i111] * x_d;

        c0 = c00 * (1 - y_d) + c10 * y_d;
        c1 = c01 * (1 - y_d) + c11 * y_d;

        ret.y = (c0 * (1 - z_d) + c1 * z_d);

        c00 = w[i000] * (1 - x_d) + w[i100] * x_d;
        c10 = w[i010] * (1 - x_d) + w[i110] * x_d;
        c01 = w[i001] * (1 - x_d) + w[i101] * x_d;
        c11 = w[i011] * (1 - x_d) + w[i111] * x_d;

        c0 = c00 * (1 - y_d) + c10 * y_d;
        c1 = c01 * (1 - y_d) + c11 * y_d;

        ret.z = (c0 * (1 - z_d) + c1 * z_d);

        return ret;
    }


    __host__ __device__
    float3 get_velocity_clamp(float x, float y, float z, float t) const {
        float3 ret;
        size_t x0, x1;
        size_t y0, y1;
        size_t z0, z1;

        // clamping logic ----------
        if (x <= 0) {
            x0 = 0;
            x1 = 0;
        } else if (x >= num_x-1) {
            x0 = num_x-1;
            x1 = num_x-1;
        } else {
            x0 = (size_t)x;
            x1 = x+1;
        }

        if (y <= 0) {
            y0 = 0;
            y1 = 0;
        } else if (y >= num_y-1) {
            y0 = num_y-1;
            y1 = num_y-1;
        } else {
            y0 = (size_t)y;
            y1 = y+1;
        }

        if (z <= 0) {
            z0 = 0;
            z1 = 0;
        } else if (z >= num_z-1) {
            z0 = num_z-1;
            z1 = num_z-1;
        } else {
            z0 = (size_t)z;
            z1 = z+1;
        }

        // distance from actual point to sampled point index
        float x_d = x - x0;
        float y_d = y - y0;
        float z_d = z - z0;

        size_t i000 = get_index(x0, y0, z0, t);
        size_t i100 = get_index(x1, y1, z0, t);
        size_t i010 = get_index(x0, y1, z0, t);
        size_t i110 = get_index(x1, y1, z0, t);
        size_t i001 = get_index(x0, y0, z1, t);
        size_t i101 = get_index(x1, y0, z1, t);
        size_t i011 = get_index(x0, y1, z1, t);
        size_t i111 = get_index(x1, y1, z1, t);

        float c00 = u[i000] * (1 - x_d) + u[i100] * x_d;
        float c10 = u[i010] * (1 - x_d) + u[i110] * x_d;
        float c01 = u[i001] * (1 - x_d) + u[i101] * x_d;
        float c11 = u[i011] * (1 - x_d) + u[i111] * x_d;

        float c0 = c00 * (1 - y_d) + c10 * y_d;
        float c1 = c01 * (1 - y_d) + c11 * y_d;

        ret.x = (c0 * (1 - z_d) + c1 * z_d);

        c00 = v[i000] * (1 - x_d) + v[i100] * x_d;
        c10 = v[i010] * (1 - x_d) + v[i110] * x_d;
        c01 = v[i001] * (1 - x_d) + v[i101] * x_d;
        c11 = v[i011] * (1 - x_d) + v[i111] * x_d;

        c0 = c00 * (1 - y_d) + c10 * y_d;
        c1 = c01 * (1 - y_d) + c11 * y_d;

        ret.y = (c0 * (1 - z_d) + c1 * z_d);

        c00 = w[i000] * (1 - x_d) + w[i100] * x_d;
        c10 = w[i010] * (1 - x_d) + w[i110] * x_d;
        c01 = w[i001] * (1 - x_d) + w[i101] * x_d;
        c11 = w[i011] * (1 - x_d) + w[i111] * x_d;

        c0 = c00 * (1 - y_d) + c10 * y_d;
        c1 = c01 * (1 - y_d) + c11 * y_d;

        ret.z = (c0 * (1 - z_d) + c1 * z_d);

        return ret;
    }


    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple tup) const
    {
        float &x = thrust::get<0>(tup);
        float &y = thrust::get<1>(tup);
        float &z = thrust::get<2>(tup);
        int   &birthtime = thrust::get<3>(tup);
        bool  &has_deposited = thrust::get<4>(tup);

        if (birthtime > t)
            return;
        if (has_deposited)
            return;

        float3 vel_0 = get_velocity_clamp(x, y, z, t);

        // first guess position P1
        float x1 = x + vel_0.x;
        float y1 = y + vel_0.y;
        float z1 = z + vel_0.z;
        float3 vel_1 = get_velocity_clamp(x1, y1, z1, t+1);

        x += 0.5f * vel_0.x + 0.5f * vel_1.x;
        y += 0.5f * vel_0.y + 0.5f * vel_1.y;
        z += 0.5f * vel_0.z + 0.5f * vel_1.z;
    }
};


template <typename Particles, typename WindData>
void advect_original_generic(Particles &p, WindData &wd, float t)
{
    thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(
            p.pos_x.begin(),
            p.pos_y.begin(), 
            p.pos_z.begin(),
            p.birthtime.begin(),
            p.has_deposited.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(
            p.pos_x.end(),
            p.pos_y.end(),
            p.pos_z.end(),
            p.birthtime.end(),
            p.has_deposited.end())),
        AdvectOriginalFunctor(wd, t)
    );
}


struct AdvectOriginalWindTextureFunctor {
    AdvectOriginalWindTextureFunctor(WindDataTextureMemoryAccessor accessor, float t_)
        : t(t_), get_wind(accessor)
    { }

    const float t;
    WindDataTextureMemoryAccessor get_wind;

    template <typename Tuple>
    __device__
    void operator()(Tuple tup) const
    {
        float &x             = thrust::get<0>(tup);
        float &y             = thrust::get<1>(tup);
        float &z             = thrust::get<2>(tup);
        int   &birthtime     = thrust::get<3>(tup);
        bool  &has_deposited = thrust::get<4>(tup);

        if (birthtime > t)
            return;
        if (has_deposited)
            return;

        float4 vel_0 = get_wind(x, y, z, t);

        // first guess position P1
        float x1 = x + vel_0.x;
        float y1 = y + vel_0.y;
        float z1 = z + vel_0.z;

        float4 vel_1 = get_wind(x1, y1, z1, t);

        x += 0.5f * vel_0.x + 0.5f * vel_1.x;
        y += 0.5f * vel_0.y + 0.5f * vel_1.y;
        z += 0.5f * vel_0.z + 0.5f * vel_1.z;
    }
};

}


void advect_original(ParticleSetThrustHost &particles, const WindDataThrustHost &wind, float t)
{
    advect_original_generic(particles, wind, t);
}

void advect_original(ParticleSetThrustDevice &particles, const WindDataThrustDevice &wind, float t)
{
    advect_original_generic(particles, wind, t);
}



void advect_original(ParticleSetThrustDevice &particles, WindDataTextureMemory &wind, float t)
{
    wind.set_current_t(t);

    thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(
            particles.pos_x.begin(),
            particles.pos_y.begin(), 
            particles.pos_z.begin(),
            particles.birthtime.begin(),
            particles.has_deposited.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(
            particles.pos_x.end(),
            particles.pos_y.end(),
            particles.pos_z.end(),
            particles.birthtime.end(),
            particles.has_deposited.end())),
        AdvectOriginalWindTextureFunctor(wind.get_accessor(), t)
    );
}

